#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //for intellisense
#include "" //for intellisense

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "vec3.h"

#define PI 3.1415926536f


struct qwe {
	int a;
	int b;
	vec3 c;

	__device__ void Test() {
		a++;
	}

};

__global__ void cuda_kernel_texture_2d(qwe q, unsigned char* surface, int width, int height, size_t pitch, float* spheres, int num_sphere)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;

	if (x >= width || y >= height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)(surface + y * pitch) + 4 * x;

	pixel[0] = 0.0 * x / width;
	pixel[1] = 0.0 * y / height; // green
	pixel[2] = spheres[3]; // blue
	pixel[3] = 1; // alpha

	q.Test();
	vec3 a = vec3();
	a.cross(a);
	a.dot(a + 1);

	//float c = a.length();

	pixel[0] = q.c.x;
}

extern "C"
void cuda_texture_2d(void* surface, int width, int height, size_t pitch, float* spheres, int num_sphere)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);

	qwe q;
	q.a = 1;
	q.b = 1;
	q.c = { 1,0,1 };

	cuda_kernel_texture_2d << <Dg, Db >> > (q, (unsigned char*)surface, width, height, pitch, spheres, num_sphere);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}


__global__ void cuda_kernel_copy_colors(unsigned char* surface, int width, int height, size_t pitch, float* colors)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;
	float* color;

	if (x >= width || y >= height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)(surface + y * pitch) + 4 * x;
	color = colors + y + x * height*4;

	pixel[0] = color[0];
	pixel[1] = color[1];
	pixel[2] = color[2];
	pixel[3] = color[3];
}

extern "C"
void cuda_copy_colors(void* surface, int width, int height, size_t pitch, float* colors)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);


	cuda_kernel_copy_colors << <Dg, Db >> > ( (unsigned char*)surface, width, height, pitch, colors);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}
