#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //for intellisense
#include "" //for intellisense

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <math.h>

#include "vec3.h"
#include "Sphere.h"
#include "Scene.h"
#include "Screen.h"
#include <algorithm>

#define PI 3.1415926536f
#define FULL_MASK 0xffffffff

#define DIFFUSE 0.6
#define AMBIENT 0.2


__device__ vec3 getColor(vec3& p, vec3& n, vec3& c) {
	vec3 lightPos = vec3(0, 5, 0);

	vec3 toLight = (lightPos - p).norm();
	float diff = fmaxf(toLight.dot(n), 0);
	return c * (diff * DIFFUSE + AMBIENT);
}

__device__ Ray castScreenRay(CameraData& c, int& x, int& y, int& width, int& height)
{
	//r=1
	float xAngle = c.fov * (1.0f * x / width - 0.5f);
	float yAngle = c.fov * (1.0f * y / height - 0.5f);

	vec3 dir = (c.forward + c.right * sinf(xAngle) * c.aspect + c.up * sinf(yAngle)).norm();

	return Ray(c.position, dir);
}

__device__ float findIntersection(Sphere& s, Ray& ray) {
	float b = (2 * (ray.origin.x - s.position.x) * ray.direction.x) + (2 * (ray.origin.y - s.position.y) * ray.direction.y) + (2 * (ray.origin.z - s.position.z) * ray.direction.z);
	float c = pow(ray.origin.x - s.position.x, 2) + pow(ray.origin.y - s.position.y, 2) + pow(ray.origin.z - s.position.z, 2) - (s.radius * s.radius);

	float dist = -1;
	float discriminant = b * b - 4 * c;
	if (discriminant >= 0) {
		float tmp = sqrtf(discriminant) ;
		dist = ((-b - tmp) / 2) - 0.000001;
		if (dist < 0)
			dist = ((-b + tmp) / 2) - 0.000001;
	}

	return dist;

}
__global__ void cuda_kernel_texture_2dx(Screen screen, Scene scene)
{
	int si = threadIdx.x;
	int x = threadIdx.y + blockDim.x * blockIdx.x;
	int y = blockIdx.y;
	float* pixel;

	if (x >= screen.width || y >= screen.height) return;

	pixel = (float*)((char*)screen.surface + y * screen.pitch) + 4 * x;

	Ray ray = Ray();// scene.cam->CastScreenRay(x, y);

	//__shared__ Sphere spheres[32];
	//spheres[si] = scene.spheres[si];
	//int idx = si;
	//float dist = spheres[si].findIntersection(ray);


	//if (__any_sync(FULL_MASK, dist > 0))
	//{
	//	for (int offset = 1; offset < 32; offset <<= 2) {
	//		float d = __shfl_down_sync(0xFFFFFFFF, dist, offset, 32);
	//		int i = __shfl_down_sync(0xFFFFFFFF, idx, offset, 32);
	//		if (dist < 0 || (d > -1 && d < dist))
	//		{
	//			dist = d;
	//			idx = i;
	//		}
	//	}
	//}

	//if (si == 0)
	//{
	//	if (dist != -1)
	//	{
	//		vec3 p = ray.getPointAt(dist);
	//		Sphere sphere = spheres[5];
	//		vec3 col = scene.light->getColor(scene.cam, p, sphere.getNormalAt(p), sphere.color);
	//		pixel[0] = col.x;// 0.0 * x / width;
	//		pixel[1] = col.y;// 0.0 * y / height; // green
	//		pixel[2] = col.z; // blue
	//	}
	//	else {
	//		pixel[0] = 0.0f;
	//		pixel[1] = 1.0f;
	//		pixel[2] = 0.0f; // blue
	//	}
	//}
}


__global__ void cuda_kernel_texture_2d(Screen screen, Scene scene)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;

	if (x >= screen.width || y >= screen.height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)((char*)screen.surface + y * screen.pitch) + 4 * x;

	Ray ray = castScreenRay(scene.cam, x, y, screen.width, screen.height);

	pixel[3] = 1.0f; // alpha

	float minDist = 10000;
	int idx = -1;

	for (int i = 0; i < scene.sphere_num; i++)
	{
		Sphere sphere = scene.spheres[i];
		float dist = findIntersection(sphere, ray);
		if (dist > 0 && dist < minDist) {
			minDist = dist;
			idx = i;
		}
	}
	if (idx >= 0)
	{
		vec3 p = ray.getPointAt(minDist);
		Sphere sphere = scene.spheres[idx];
		//vec3 col = scene.light->getColor(scene.cam, p, sphere.getNormalAt(p), sphere.color);
		vec3 n = p - sphere.position;
		vec3 col = getColor(p,n , sphere.color);
		pixel[0] = col.x;// 0.0 * x / width;
		pixel[1] = col.y;// 0.0 * y / height; // green
		pixel[2] = col.z; // blue
	}
	else {
		pixel[0] = 0.0f;
		pixel[1] = 1.0f;
		pixel[2] = 0.0f; // blue

	}
}

extern "C"
void cuda_texture_2dx(Screen screen, Scene scene)
{
	hipError_t error = hipSuccess;

	dim3 gridSize = dim3(screen.width / 32, screen.height, 1);
	//dim3 gridSize = dim3(screen.width / 32, 100, 1);
	dim3 blockSize = dim3(32, 32, 1);

	cuda_kernel_texture_2d << < gridSize, blockSize >> > (screen, scene);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}


extern "C"
void cuda_texture_2d(Screen screen, Scene scene)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((screen.width + Db.x - 1) / Db.x, (screen.width + Db.y - 1) / Db.y);

	cuda_kernel_texture_2d << <Dg, Db >> > (screen, scene);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}
