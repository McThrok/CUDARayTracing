#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //for intellisense
#include "" //for intellisense

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "vec3.h"
#include "Sphere.h"
#include "Scene.h"
#include "Screen.h"
#include <algorithm>

#define PI 3.1415926536f


__global__ void cuda_kernel_texture_2d(Screen screen, Scene scene)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;

	if (x >= screen.width || y >= screen.height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)((char*)screen.surface + y * screen.pitch) + 4 * x;

	Ray ray = scene.cam.CastScreenRay(x, y);

	pixel[3] = 1.0f; // alpha
	Sphere sphere = scene.spheres[0];
	float dist = sphere.findIntersection(ray);
	if (dist > 0)
	{
		vec3 p = ray.getPointAt(dist);
		vec3 col = scene.light.getColor(scene.cam, p, sphere.getNormalAt(p),sphere.color);
		pixel[0] = col.x;// 0.0 * x / width;
		pixel[1] = col.y;// 0.0 * y / height; // green
		pixel[2] = col.z; // blue
	}
	else {
		pixel[0] = 0.0f;// 0.0 * x / width;
		pixel[1] = 1.0f;// 0.0 * y / height; // green
		pixel[2] = 0.0f; // blue

	}

}

extern "C"
void cuda_texture_2d(Screen screen, Scene scene)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((screen.width + Db.x - 1) / Db.x, (screen.width + Db.y - 1) / Db.y);

	cuda_kernel_texture_2d << <Dg, Db >> > (screen, scene);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}


__global__ void cuda_kernel_copy_colors(unsigned char* surface, int width, int height, size_t pitch, float* colors)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;
	float* color;

	if (x >= width || y >= height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)(surface + y * pitch) + 4 * x;
	color = colors + y + x * height * 4;

	pixel[0] = color[0];
	pixel[1] = color[1];
	pixel[2] = color[2];
	pixel[3] = color[3];
}

extern "C"
void cuda_copy_colors(void* surface, int width, int height, size_t pitch, float* colors)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);


	cuda_kernel_copy_colors << <Dg, Db >> > ((unsigned char*)surface, width, height, pitch, colors);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}
