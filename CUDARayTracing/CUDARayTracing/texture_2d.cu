#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //for intellisense
#include "" //for intellisense

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "vec3.h"
#include "Sphere.h"
#include "Scene.h"
#include "Screen.h"

#define PI 3.1415926536f


__global__ void cuda_kernel_texture_2d(Screen screen, Scene scene)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;

	if (x >= screen.width || y >= screen.height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)((char*)screen.surface + y * screen.pitch) + 4 * x;

	Ray ray = scene.cam.CastScreenRay(x, y);

	pixel[3] = 1.0f; // alpha
	if (scene.spheres[0].findIntersection(ray) > 0)
	{
		pixel[0] = 1.0f;// 0.0 * x / width;
		pixel[1] = 0.0f;// 0.0 * y / height; // green
		pixel[2] = 0.0f; // blue
	}
	else {
		pixel[0] = 0.0f;// 0.0 * x / width;
		pixel[1] = 1.0f;// 0.0 * y / height; // green
		pixel[2] = 0.0f; // blue

	}

}

extern "C"
void cuda_texture_2d(Screen screen, Scene scene)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((screen.width + Db.x - 1) / Db.x, (screen.width + Db.y - 1) / Db.y);

	cuda_kernel_texture_2d << <Dg, Db >> > (screen, scene);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}


__global__ void cuda_kernel_copy_colors(unsigned char* surface, int width, int height, size_t pitch, float* colors)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;
	float* color;

	if (x >= width || y >= height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)(surface + y * pitch) + 4 * x;
	color = colors + y + x * height * 4;

	pixel[0] = color[0];
	pixel[1] = color[1];
	pixel[2] = color[2];
	pixel[3] = color[3];
}

extern "C"
void cuda_copy_colors(void* surface, int width, int height, size_t pitch, float* colors)
{
	hipError_t error = hipSuccess;

	dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Dg = dim3((width + Db.x - 1) / Db.x, (height + Db.y - 1) / Db.y);


	cuda_kernel_copy_colors << <Dg, Db >> > ((unsigned char*)surface, width, height, pitch, colors);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}
