#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h" //for intellisense
#include "" //for intellisense

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <math.h>

#include "vec3.h"
#include "Scene.h"
#include "Screen.h"
#include <algorithm>

#define PI 3.1415926536f

#define DIFFUSE 0.5
#define AMBIENT 0.2


__device__ vec3 getSphereColor(vec3& position, vec3& color, vec3& point) {
	vec3 toLight;
	float diff;

	vec3 normal = point - position;
	vec3 c = vec3(AMBIENT, AMBIENT, AMBIENT);

	toLight = (vec3( 1, 0, -0.5)).norm();
	diff = fmaxf(toLight.dot(normal), 0);
	c += (diff * DIFFUSE);


	toLight = (vec3(-1, 0, -0.5)).norm();
	diff = fmaxf(toLight.dot(normal), 0);
	c += (diff * DIFFUSE);

	toLight = (vec3( 0, 1, -0.5)).norm();
	diff = fmaxf(toLight.dot(normal), 0);
	c += (diff * DIFFUSE);

	return color * c;
}

__device__ vec3 castScreenRay(CameraData& c, int& x, int& y, int& width, int& height) {
	//r=1
	float xAngle = c.fov * (1.0f * x / width - 0.5f);
	float yAngle = c.fov * (1.0f * y / height - 0.5f);

	return vec3(c.forward + c.right * sinf(xAngle) * c.aspect + c.up * sinf(yAngle)).norm();
}

__device__ float findIntersection(vec3& pos, float& r, vec3& rayOrigin, vec3& rayDir) {
	vec3 v(rayOrigin.x - pos.x, rayOrigin.y - pos.y, rayOrigin.z - pos.z);
	float b = (2 * v.x * rayDir.x) + (2 * v.y * rayDir.y) + (2 * v.z * rayDir.z);
	float c = v.dot(v) - (r * r);

	float dist = -1;
	float discriminant = b * b - 4 * c;
	if (discriminant >= 0) {
		float tmp = sqrtf(discriminant);
		dist = ((-b - tmp) / 2) - 0.000001f;
		if (dist < 0)
			dist = ((-b + tmp) / 2) - 0.000001f;
	}

	return dist;
}

__global__ void draw(Screen screen, Scene scene)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	float* pixel;

	if (x >= screen.width || y >= screen.height) return;

	// get a pointer to the pixel at (x,y)
	pixel = (float*)((char*)screen.surface + y * screen.pitch) + 4 * x;

	vec3 rayDir = castScreenRay(scene.cam, x, y, screen.width, screen.height);
	vec3 rayOrigin = scene.cam.position;

	pixel[3] = 1.0f; // alpha

	float minDist = 10000;
	int idx = -1;

	for (int i = 0; i < scene.sphere_num; i++)
	{
		float dist = findIntersection(scene.position[i], scene.radius[i], rayOrigin, rayDir);
		if (dist > 0 && dist < minDist) {
			minDist = dist;
			idx = i;
		}
	}

	if (idx >= 0)
	{
		vec3 p = rayOrigin + rayDir * minDist;
		vec3 col = getSphereColor(scene.position[idx], scene.color[idx], p);

		pixel[0] = col.x;
		pixel[1] = col.y;
		pixel[2] = col.z;
	}
	else {
		pixel[0] = 0.0;
		pixel[1] = 0.0f;
		pixel[2] = 0.0f;

	}
}

extern "C"
void execute_kernel(Screen screen, Scene scene)
{
	hipError_t error = hipSuccess;

	//dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
	dim3 Db = dim3(32, 16);
	dim3 Dg = dim3(screen.width / Db.x, screen.height / Db.y);

	draw << <Dg, Db >> > (screen, scene);

	error = hipGetLastError();

	if (error != hipSuccess)
	{
		printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
	}
}
